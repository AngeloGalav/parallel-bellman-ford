#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 512 // threads per block
#define THREAD_ID (blockIdx.x * blockDim.x + threadIdx.x);
#define N_OF_BLOCKS(N) ((N + BLOCK_SIZE - 1) / BLOCK_SIZE) // total n of blocks

#define cudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define cudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )

//  Different implementation of cuda_gettime depending on
//  the OS used by the user, as Windows is not POSIX compliant
#ifdef _WIN32
#include <windows.h>
#define MS_PER_SEC      1000ULL     // MS = milliseconds
#define US_PER_MS       1000ULL     // US = microseconds
#define HNS_PER_US      10ULL       // HNS = hundred-nanoseconds (e.g., 1 hns = 100 ns)
#define NS_PER_US       1000ULL

#define HNS_PER_SEC     (MS_PER_SEC * US_PER_MS * HNS_PER_US)
#define NS_PER_HNS      (100ULL)    // NS = nanoseconds
#define NS_PER_SEC      (MS_PER_SEC * US_PER_MS * NS_PER_US)

double cuda_gettime(void)
{
    struct timespec ts;
    static LARGE_INTEGER ticksPerSec;
    LARGE_INTEGER ticks;

    if (!ticksPerSec.QuadPart) {
        QueryPerformanceFrequency(&ticksPerSec);
        if (!ticksPerSec.QuadPart) {
            errno = ENOTSUP;
            return -1;
        }
    }

    QueryPerformanceCounter(&ticks);

    ts.tv_sec = (long)(ticks.QuadPart / ticksPerSec.QuadPart);
    ts.tv_nsec = (long)(((ticks.QuadPart % ticksPerSec.QuadPart) * NS_PER_SEC) / ticksPerSec.QuadPart);
    return ts.tv_sec + (double)ts.tv_nsec / 1e9;
}
#elif __unix__
double cuda_gettime( void )
{
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts );
    return ts.tv_sec + (double)ts.tv_nsec / 1e9;
}
#endif

typedef struct node {
    int id;
    int cost;
    struct node *next;
} Node;

typedef struct graph {
    int V, E;
    Node **nodes;
} Graph;

void copyLinkedList(Node** d_nodes, Node* h_nodeList, int index) {
    Node* d_current = d_nodes[index];
    Node* hd = h_nodeList;
    while (hd != NULL) {
        // Allocate memory for each node on the device
        Node* d_newNode;
        hipMalloc((void**)&d_newNode, sizeof(Node));
        hipMemcpy(d_newNode, hd, sizeof(Node), hipMemcpyHostToDevice);

        // Link the nodes
        if (d_current == NULL) {
            d_nodes[index] = d_newNode;
        } else {
            Node* d_lastNode = d_current;
            while (d_lastNode->next != NULL) {
                d_lastNode = d_lastNode->next;
            }
            d_lastNode->next = d_newNode;
        }

        hd = hd->next;
    }
}

void freeGraphMemory(Graph* d_graph, const Graph& h_graph) {
    // Free each linked list
    for (int i = 0; i < h_graph.V; i++) {
        Node* node = d_graph->nodes[i];
        while (node != NULL) {
            Node* next = node->next;
            hipFree(node);
            node = next;
        }
    }
    // Free nodes array
    hipFree(d_graph->nodes);
}

void copyGraphToDevice(const Graph& h_graph, Graph* d_graph) {
    hipMemcpy(d_graph, &h_graph, sizeof(Graph), hipMemcpyHostToDevice);

    // Copy each linked list
    for (int i = 0; i < h_graph.V; i++) {
        Node* h_nodeList = h_graph.nodes[i];
        Node* d_nodeList = NULL;
        copyLinkedList(d_graph->nodes, h_nodeList, i);
    }
}

__global__ void initDistArray(int* dist, int V){
    int i = THREAD_ID;
    if (i < V) {
        dist[i] = INT_MAX;
    }

    if (i == 0) dist[i] = 0;
}

__global__ void checkNegative(int* dist, int V, Graph* graph, int &neg_check){
    int i = THREAD_ID;
    if (i < V) {
        int u = i;
        Node* hd = graph->nodes[0]->next;

        while (hd != NULL) {
            int v = hd->id;
            int weight = hd->cost;

            // If negative cycle is detected, simply return
            if (dist[u] != INT_MAX && dist[u] + weight < dist[v]) {
                printf("Graph contains negative weight cycle\n");
                neg_check = 1;
                return;
            }

            hd = hd->next;
        }
    }
}

__global__ void relaxationStep(int* dist, int V, Graph* graph){
    int i = THREAD_ID;
    if (i < V) {
        int u = i;
        Node* hd = graph->nodes[u]->next;
        while (hd != NULL) {
            int v = hd->id;
            int weight = hd->cost;
            if (dist[u] != INT_MAX && (dist[u] + weight) < dist[v]) {
                int upd = dist[u];
                __syncthreads();
                dist[v] = upd + weight;
            }

            hd = hd->next;
        }
    }
}


int *BellmanFord(Graph *graph, int src);
void printArr(int dist[], int n);
Graph *initGraph(int V, int E);
void addEdge(Graph *graph, int src, int dest, int cost, int bidirectional);
void printInfoToFile(char *graph_file, double total_time, int threads);
Graph *createGraphFromFile(char *filename);
void printGraph(Graph *graph);
void printEdgesOfNode(Node *node);

int main(int argc, char *argv[]) {
    // char *n_threads_s = argv[1];
    // char *graph_file = argv[2];

    // // doing a background check on these guys
    // // if (n_threads_s == NULL) {
    // //     printf("ERROR: No threads inputted.\n");
    // //     return -1;
    // // }

    // if (graph_file == NULL) {
    //     printf("ERROR: No graph file inputted.\n");
    //     return -1;
    // }

    // // setting the number of threads
    // int n_threads = atoi(n_threads_s);

    // generating the graph
    const char* filename = "graphs/graph_5.txt";
    char graph_filename[100];
    strcpy(graph_filename, filename);

    Graph *graph = createGraphFromFile(graph_filename);
    if (graph == NULL) {
        printf("Error creating graph\n");
        return -1;
    };

    double time_start, time_end;

    time_start = cuda_gettime();
    int *dist_result;
    dist_result = BellmanFord(graph, 0);
    hipDeviceSynchronize();
    time_end = cuda_gettime();

    double total_time = time_end - time_start;

    // printing the distance array (i.e. the result)
    // printArr(dist_result, graph->V);

    printf("\n");

    printf("Total execution time: %f seconds\n", total_time);
    // printInfoToFile(graph_file, total_time, n_threads);

    return 0;
}

Graph *createGraphFromFile(char *filename) {
    FILE *file = fopen(filename, "r");
    if (file == NULL) {
        printf("Error opening file.\n");
        return NULL;
    }

    int V, E;
    fscanf(file, "%d %d", &V, &E);
    Graph *graph = initGraph(V, E);

    for (int i = 0; i < E; i++) {
        int u, v, weight;
        fscanf(file, "%d %d %d", &u, &v, &weight);
        addEdge(graph, u, v, weight, 1);
    }

    fclose(file);
    return graph;
}

void printInfoToFile(char *graph_file, double total_time, int threads) {
    // Define the file path
    char file_path[256];
    snprintf(file_path, sizeof(file_path), "results/cuda.csv");

    FILE *file = fopen(file_path, "a");
    if (!file) {
        fprintf(stderr,
                "Failed to open file: %s, creating file on WD instead...\n",
                file_path);
        snprintf(file_path, sizeof(file_path), "cuda.csv");
        file = fopen(file_path, "a");
        if (!file) {
            fprintf(stderr, "Failed to open file again, aborting.");
            return;
        }
        printf("File created successfully!\n");
    }

    fprintf(file, "%s,%d,%.6f\n", graph_file, threads, total_time);
    fclose(file);
}

int *BellmanFord(Graph *graph, int src) {
    int V = graph->V;

    Graph* graph_gpu;
    hipMalloc(&graph_gpu, sizeof(Graph));
    hipMalloc(&(graph_gpu->nodes), (V * sizeof(Node *)));
    copyGraphToDevice(*graph, graph_gpu);

    int *dist;
    hipMalloc(&dist, sizeof(int) * V);

    initDistArray<<<N_OF_BLOCKS(V), BLOCK_SIZE>>>(dist, V);
    hipDeviceSynchronize();

    relaxationStep<<<N_OF_BLOCKS(V), BLOCK_SIZE>>>(dist, V, graph_gpu);
    hipDeviceSynchronize();

    int* neg_check;
    hipMalloc(&neg_check, sizeof(int));
    checkNegative<<<N_OF_BLOCKS(V), BLOCK_SIZE>>>(dist, V, graph_gpu, *neg_check);
    hipDeviceSynchronize();

    if (neg_check) return NULL;
    return dist;
}

void printArr(int dist[], int n) {
    printf("Vertex  |  Distance from Source\n");
    for (int i = 0; i < n; ++i) printf("%d \t\t %d\n", i, dist[i]);
}

//
// Graph handling functions
//
//

Graph *initGraph(int V, int E) {
    Graph *graph = (Graph *)malloc(sizeof(Graph));
    graph->V = V;
    graph->E = E;
    graph->nodes = (Node **)malloc(V * sizeof(Node *));

    for (int i = 0; i < V; i++) {
        graph->nodes[i] = (Node*)malloc(sizeof(Node));
        graph->nodes[i]->id = i;
        graph->nodes[i]->next = NULL;
    }

    return graph;
}

void addEdge(Graph *graph, int src, int dest, int cost, int bidirectional) {
    Node *hd = graph->nodes[src];
    while (hd->next != NULL) {
        hd = hd->next;
    }
    hd->next = (Node*) malloc(sizeof(Node));
    hd->next->id = dest;
    hd->next->cost = cost;
    hd->next->next = NULL;

    if (bidirectional) addEdge(graph, dest, src, cost, 0);
}

void printGraph(Graph *graph) {
    printf("Graph edges:\n");
    for (int i = 0; i < graph->V; ++i) {
        printf("info of %d: ", i);
        printEdgesOfNode(graph->nodes[i]);
    }
}

void printEdgesOfNode(Node *node) {
    Node *hd = node;
    printf("Node %d is attached to nodes", node->id);
    while (hd->next != NULL) {
        hd = hd->next;
        printf(" %d,", hd->id);
    }
    printf(" - END\n");
}