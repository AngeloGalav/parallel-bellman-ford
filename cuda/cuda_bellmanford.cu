#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define cudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define cudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )

//  Different implementation of cuda_gettime depending on
//  the OS used by the user, as Windows is not POSIX compliant
#ifdef _WIN32
#include <windows.h>
#define MS_PER_SEC      1000ULL     // MS = milliseconds
#define US_PER_MS       1000ULL     // US = microseconds
#define HNS_PER_US      10ULL       // HNS = hundred-nanoseconds (e.g., 1 hns = 100 ns)
#define NS_PER_US       1000ULL

#define HNS_PER_SEC     (MS_PER_SEC * US_PER_MS * HNS_PER_US)
#define NS_PER_HNS      (100ULL)    // NS = nanoseconds
#define NS_PER_SEC      (MS_PER_SEC * US_PER_MS * NS_PER_US)

double cuda_gettime(void)
{
    struct timespec ts;
    static LARGE_INTEGER ticksPerSec;
    LARGE_INTEGER ticks;

    if (!ticksPerSec.QuadPart) {
        QueryPerformanceFrequency(&ticksPerSec);
        if (!ticksPerSec.QuadPart) {
            errno = ENOTSUP;
            return -1;
        }
    }

    QueryPerformanceCounter(&ticks);

    ts.tv_sec = (long)(ticks.QuadPart / ticksPerSec.QuadPart);
    ts.tv_nsec = (long)(((ticks.QuadPart % ticksPerSec.QuadPart) * NS_PER_SEC) / ticksPerSec.QuadPart);
    return ts.tv_sec + (double)ts.tv_nsec / 1e9;
}
#elif __unix__
double cuda_gettime( void )
{
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts );
    return ts.tv_sec + (double)ts.tv_nsec / 1e9;
}
#endif

typedef struct node {
    int id;
    int cost;
    struct node *next;
} Node;

typedef struct graph {
    int V, E;
    Node **nodes;
} Graph;

__global__ void initDistArray(){

}

int *BellmanFord(Graph *graph, int src);
void printArr(int dist[], int n);
Graph *initGraph(int V, int E);
void addEdge(Graph *graph, int src, int dest, int cost, int bidirectional);
void printInfoToFile(char *graph_file, double total_time, int threads);
Graph *createGraphFromFile(char *filename);
void printGraph(Graph *graph);
void printEdgesOfNode(Node *node);

int main(int argc, char *argv[]) {
    // char *n_threads_s = argv[1];
    // char *graph_file = argv[2];

    // // doing a background check on these guys
    // // if (n_threads_s == NULL) {
    // //     printf("ERROR: No threads inputted.\n");
    // //     return -1;
    // // }

    // if (graph_file == NULL) {
    //     printf("ERROR: No graph file inputted.\n");
    //     return -1;
    // }

    // // setting the number of threads
    // int n_threads = atoi(n_threads_s);

    // generating the graph
    const char* filename = "graphs/graph_5.txt";
    char graph_filename[100];
    strcpy(graph_filename, filename);

    Graph *graph = createGraphFromFile(graph_filename);
    if (graph == NULL) {
        printf("Error creating graph\n");
        return -1;
    };

    double time_start, time_end;

    time_start = cuda_gettime();
    int *dist_result;
    dist_result = BellmanFord(graph, 0);
    time_end = cuda_gettime();

    double total_time = time_end - time_start;

    // printing the distance array (i.e. the result)
    // printArr(dist_result, graph->V);

    printf("\n");

    printf("Total execution time: %f seconds\n", total_time);
    // printInfoToFile(graph_file, total_time, n_threads);

    return 0;
}

Graph *createGraphFromFile(char *filename) {
    FILE *file = fopen(filename, "r");
    if (file == NULL) {
        printf("Error opening file.\n");
        return NULL;
    }

    int V, E;
    fscanf(file, "%d %d", &V, &E);
    Graph *graph = initGraph(V, E);

    for (int i = 0; i < E; i++) {
        int u, v, weight;
        fscanf(file, "%d %d %d", &u, &v, &weight);
        addEdge(graph, u, v, weight, 1);
    }

    fclose(file);
    return graph;
}

void printInfoToFile(char *graph_file, double total_time, int threads) {
    // Define the file path
    char file_path[256];
    snprintf(file_path, sizeof(file_path), "results/cuda.csv");

    FILE *file = fopen(file_path, "a");
    if (!file) {
        fprintf(stderr,
                "Failed to open file: %s, creating file on WD instead...\n",
                file_path);
        snprintf(file_path, sizeof(file_path), "cuda.csv");
        file = fopen(file_path, "a");
        if (!file) {
            fprintf(stderr, "Failed to open file again, aborting.");
            return;
        }
        printf("File created successfully!\n");
    }

    fprintf(file, "%s,%d,%.6f\n", graph_file, threads, total_time);
    fclose(file);
}

int *BellmanFord(Graph *graph, int src) {
    int V = graph->V;
    int *dist = (int*)malloc(sizeof(int) * V);
    if (dist == NULL) {
        perror("Failed to allocate memory");
        return NULL;
    }

    // declaring variables here in order for the scoping to work
    int i, j, u, v, weight;
    Node *hd;

    for (i = 0; i < V; i++) {
        dist[i] = INT_MAX;
    }

    dist[src] = 0;

    // compute distance array
    for (i = 1; i < V; i++) {
        for (j = 0; j < V; j++) {
            u = j;
            hd = graph->nodes[u]->next;
            while (hd != NULL) {
                v = hd->id;
                weight = hd->cost;

                if (dist[u] != INT_MAX && (dist[u] + weight) < dist[v]) {
                    dist[v] = dist[u] + weight;
                }

                hd = hd->next;
            }
        }
    }

    int neg_check = 0;

    //  if graph still has a shorter path, then there's a negative cycle
    for (i = 0; i < V; i++) {
        u = i;
        hd = graph->nodes[0]->next;

        while (hd != NULL) {
            int v = hd->id;
            int weight = hd->cost;

            // If negative cycle is detected, simply return
            if (dist[u] != INT_MAX && dist[u] + weight < dist[v]) {
                printf("Graph contains negative weight cycle\n");
                neg_check = 1;
            }

            hd = hd->next;
        }
    }

    if (neg_check) return NULL;
    return dist;
}

void printArr(int dist[], int n) {
    printf("Vertex  |  Distance from Source\n");
    for (int i = 0; i < n; ++i) printf("%d \t\t %d\n", i, dist[i]);
}

//
// Graph handling functions
//
//

Graph *initGraph(int V, int E) {
    Graph *graph = (Graph *)malloc(sizeof(Graph));
    graph->V = V;
    graph->E = E;
    graph->nodes = (Node **)malloc(V * sizeof(Node *));

    for (int i = 0; i < V; i++) {
        graph->nodes[i] = (Node*)malloc(sizeof(Node));
        graph->nodes[i]->id = i;
        graph->nodes[i]->next = NULL;
    }

    return graph;
}

void addEdge(Graph *graph, int src, int dest, int cost, int bidirectional) {
    Node *hd = graph->nodes[src];
    while (hd->next != NULL) {
        hd = hd->next;
    }
    hd->next = (Node*) malloc(sizeof(Node));
    hd->next->id = dest;
    hd->next->cost = cost;
    hd->next->next = NULL;

    if (bidirectional) addEdge(graph, dest, src, cost, 0);
}

void printGraph(Graph *graph) {
    printf("Graph edges:\n");
    for (int i = 0; i < graph->V; ++i) {
        printf("info of %d: ", i);
        printEdgesOfNode(graph->nodes[i]);
    }
}

void printEdgesOfNode(Node *node) {
    Node *hd = node;
    printf("Node %d is attached to nodes", node->id);
    while (hd->next != NULL) {
        hd = hd->next;
        printf(" %d,", hd->id);
    }
    printf(" - END\n");
}