#include <hip/hip_runtime.h>
#include <limits.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#define BLOCK_SIZE 512  // threads per block
#define THREAD_ID (blockIdx.x * blockDim.x + threadIdx.x);
#define N_OF_BLOCKS(N) ((N + BLOCK_SIZE - 1) / BLOCK_SIZE)  // total n of blocks

//  Different implementation of cuda_gettime depending on
//  the OS used by the user, as Windows is not POSIX compliant
#ifdef _WIN32
#include <windows.h>
#define MS_PER_SEC 1000ULL  // MS = milliseconds
#define US_PER_MS 1000ULL   // US = microseconds
#define HNS_PER_US 10ULL    // HNS = hundred-nanoseconds (e.g., 1 hns = 100 ns)
#define NS_PER_US 1000ULL

#define HNS_PER_SEC (MS_PER_SEC * US_PER_MS * HNS_PER_US)
#define NS_PER_HNS (100ULL)  // NS = nanoseconds
#define NS_PER_SEC (MS_PER_SEC * US_PER_MS * NS_PER_US)

double cuda_gettime(void) {
    struct timespec ts;
    static LARGE_INTEGER ticksPerSec;
    LARGE_INTEGER ticks;

    if (!ticksPerSec.QuadPart) {
        QueryPerformanceFrequency(&ticksPerSec);
        if (!ticksPerSec.QuadPart) {
            errno = ENOTSUP;
            return -1;
        }
    }

    QueryPerformanceCounter(&ticks);

    ts.tv_sec = (long)(ticks.QuadPart / ticksPerSec.QuadPart);
    ts.tv_nsec = (long)(((ticks.QuadPart % ticksPerSec.QuadPart) * NS_PER_SEC) /
                        ticksPerSec.QuadPart);
    return ts.tv_sec + (double)ts.tv_nsec / 1e9;
}
#elif __unix__
double cuda_gettime(void) {
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return ts.tv_sec + (double)ts.tv_nsec / 1e9;
}
#endif

typedef struct edge {
    int u;
    int v;
    int cost;
} Edge;

typedef struct graph {
    int V, E;
    Edge *edges;
} Graph;

int *BellmanFord(Graph *graph, int src, double *time);
void printArr(int dist[], int n);
Graph *initGraph(int V, int E);
void addEdge(Graph *graph, int src, int dest, int cost, int bidirectional);
void printInfoToFile(char *graph_file, double total_time);
Graph *createGraphFromFile(char *filename, int bidirectional);

// single threaded functions in cuda
__global__ void initDistArray(int *dist, int V, int src);
__global__ void relaxationStep(int *dist, int E, Edge *edges);
__global__ void checkNegative(int *dist, int E, Edge *edges, int *neg_check);
// single threaded functions in cuda
__global__ void oneThreadInitDistArray(int *dist, int V, int src);
__global__ void oneThreadRelaxationStep(int *dist, int E, Edge *edges);
__global__ void oneThreadCheckNegative(int *dist, int E, Edge *edges, int *neg_check);

int mode; // 1 is parallel, 0 is serial

int main(int argc, char *argv[]) {
    char *graph_file = argv[1];
    char *exec_mode = argv[2];
    char *debug_flag = argv[3];

    if (exec_mode != NULL) {
        mode = atoi(exec_mode);
    } else mode = 1;

    if (graph_file == NULL) {
        printf("ERROR: No graph file inputted.\n");
        return -1;
    }

    // building the graph
    const char *filename = graph_file;
    char graph_filename[100];
    strcpy(graph_filename, filename);

    Graph *graph = createGraphFromFile(graph_filename, 1);
    if (graph == NULL) {
        printf("Error creating graph\n");
        return -1;
    };

    // distance array on GPU
    int *dist_gpu;
    // time "without" memory overhead
    double *total_time = (double *)malloc(sizeof(double));
    double time_start, time_end;

    time_start = cuda_gettime();
    dist_gpu = BellmanFord(graph, 0, total_time);
    hipDeviceSynchronize();
    time_end = cuda_gettime();

    // time with memory overhead
    double total_time_ov = time_end - time_start;
    if (dist_gpu == NULL) return -1;

    // create cpu copy of distance  array
    int *dist_cpu = (int *)malloc(sizeof(int) * graph->V);
    hipMemcpy(dist_cpu, dist_gpu, sizeof(int) * graph->V,
               hipMemcpyDeviceToHost);

    // printing the distance array (i.e. the result)
    if (debug_flag != NULL) {
        printArr(dist_cpu, graph->V);
    }
    hipFree(dist_gpu);
    free(dist_cpu);

    printf("Total execution time: %f seconds\n", *total_time);
    printf("With memory transfer overhead: %f seconds\n", total_time_ov);
    printInfoToFile(graph_file, *total_time);
    printf("\n");

    return 0;
}

int *BellmanFord(Graph *graph, int src, double *total_time) {
    int V = graph->V;
    int E = graph->E;

    // allocate gpu memory
    Edge *edges_gpu;
    hipMalloc(&edges_gpu, (E * sizeof(Edge)));
    hipMemcpy(edges_gpu, graph->edges, E * sizeof(Edge),
               hipMemcpyHostToDevice);

    int *dist;
    hipMalloc(&dist, sizeof(int) * V);

    int *neg_check_gpu;
    int neg_check;
    hipMalloc(&neg_check_gpu, sizeof(int));

    double time_start, time_end;

    // introduced time inside bellman-ford function for more precise
    // execution timing
    time_start = cuda_gettime();

    // parallel code
    if (mode) {
        initDistArray<<<N_OF_BLOCKS(V), BLOCK_SIZE>>>(dist, V, src);
        hipDeviceSynchronize();

        // relaxation step must be done V times
        for (int i = 1; i < V; i++) {
            relaxationStep<<<N_OF_BLOCKS(E), BLOCK_SIZE>>>(dist, E, edges_gpu);
        }
        hipDeviceSynchronize();

        checkNegative<<<N_OF_BLOCKS(E), BLOCK_SIZE>>>(dist, E, edges_gpu, neg_check_gpu);
        hipDeviceSynchronize();
    }
    else // serial mode
    {
        oneThreadInitDistArray<<<1, 1>>>(dist, V, src);
        hipDeviceSynchronize();

        for (int i = 1; i < V; i++) {
            oneThreadRelaxationStep<<<1, 1>>>(dist, E, edges_gpu);
        }
        hipDeviceSynchronize();

        oneThreadCheckNegative<<<1, 1>>>(dist, E, edges_gpu, neg_check_gpu);
        hipDeviceSynchronize();
    }
    hipMemcpy(&neg_check, neg_check_gpu, sizeof(int), hipMemcpyDeviceToHost);
    time_end = cuda_gettime();

    *total_time = time_end - time_start;

    hipFree(edges_gpu);
    hipFree(neg_check_gpu);
    if (neg_check) return NULL;
    return dist;
}

// parallel functions
__global__ void initDistArray(int *dist, int V, int src) {
    int i = THREAD_ID;
    if (i < V) {
        dist[i] = INT_MAX;
    }

    if (i == src) dist[i] = 0;
}

__global__ void checkNegative(int *dist, int E, Edge *edges, int *neg_check) {
    int i = THREAD_ID;
    if (i < E) {
        int u = edges[i].u;
        int v = edges[i].v;
        int weight = edges[i].cost;

        if (dist[u] != INT_MAX && dist[u] + weight < dist[v]) {
            *neg_check = 1;
        }
    }
}

__global__ void relaxationStep(int *dist, int E, Edge *edges) {
    int i = THREAD_ID;
    if (i < E) {
        int u = edges[i].u;
        int v = edges[i].v;
        int weight = edges[i].cost;

        if (dist[u] != INT_MAX && (dist[u] + weight) < dist[v]) {
            __syncthreads();
            dist[v] = dist[u] + weight;
        }
    }
}

// single thread functions
__global__ void oneThreadInitDistArray(int *dist, int V, int src) {
    for (int i = 0; i < V; i++) {
        dist[i] = INT_MAX;
    }

    dist[src] = 0;
}

__global__ void oneThreadRelaxationStep(int *dist, int E, Edge *edges) {
    for (int i = 0; i < E; i++){
        int u = edges[i].u;
        int v = edges[i].v;
        int weight = edges[i].cost;

        if (dist[u] != INT_MAX && (dist[u] + weight) < dist[v]) {
            dist[v] = dist[u] + weight;
        }
    }
}

__global__ void oneThreadCheckNegative(int *dist, int E, Edge *edges, int *neg_check) {
    for (int i = 0; i < E; i++){
        int u = edges[i].u;
        int v = edges[i].v;
        int weight = edges[i].cost;

        if (dist[u] != INT_MAX && dist[u] + weight < dist[v]) {
            *neg_check = 1;
        }
    }
}


Graph *createGraphFromFile(char *filename, int bidirectional) {
    // read file and create graph
    FILE *file = fopen(filename, "r");
    if (file == NULL) {
        printf("Error opening file.\n");
        return NULL;
    }

    int V, E;
    fscanf(file, "%d %d", &V, &E);

    // graph memory allocation
    Graph *graph = (Graph *)malloc(sizeof(Graph));
    graph->V = V;
    graph->E = E;
    if (bidirectional) graph->E = 2 * E;
    graph->edges = (Edge *)malloc(graph->E * sizeof(Edge));

    // edge definition
    for (int i = 0; i < E; i++) {
        fscanf(file, "%d %d %d", &graph->edges[i].u, &graph->edges[i].v,
               &graph->edges[i].cost);
    }

    if (bidirectional) {
        for (int i = 0; i < E; i++) {
            graph->edges[i + E].u = graph->edges[i].v;
            graph->edges[i + E].v = graph->edges[i].u;
            graph->edges[i + E].cost = graph->edges[i].cost;
        }
    }

    fclose(file);
    return graph;
}

void printInfoToFile(char *graph_file, double total_time) {
    // Define the file path
    char file_path[256];
    if (mode) snprintf(file_path, sizeof(file_path), "results/cuda.csv");
    else snprintf(file_path, sizeof(file_path), "results/cuda_serial.csv");

    FILE *file = fopen(file_path, "a");
    if (!file) {
        fprintf(stderr,
                "Failed to open file: %s, creating file on WD instead...\n",
                file_path);

        if (mode) snprintf(file_path, sizeof(file_path), "cuda.csv");
        else snprintf(file_path, sizeof(file_path), "results/cuda_serial.csv");

        file = fopen(file_path, "a");
        if (!file) {
            fprintf(stderr, "Failed to open file again, aborting.");
            return;
        }
        printf("File created successfully!\n");
    }

    fprintf(file, "%s,%.6f\n", graph_file, total_time);
    fclose(file);
}

void printArr(int dist[], int n) {
    printf("Vertex  |  Distance from Source\n");
    for (int i = 0; i < n; ++i) printf("%d \t\t %d\n", i, dist[i]);
}
